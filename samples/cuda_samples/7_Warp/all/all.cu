#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>


#include <stdio.h>

__global__ void checkValues(int* data, int* result) {
       int tid = threadIdx.x;
       int value = data[tid];

       // Check if all values in the thread warp are non-zero
       int all_non_zero = (__all(value != 0)) ? 1 : 0;

       if (tid == 0) {
               *result = all_non_zero;
       }
}

int main() {
       int num_elements = 8;
       int data[] = {1, 2, 3, 4, 5, 6, 7, 8};
       int result;
       int* d_data;
       int* d_result;

       // Allocate and copy data to the device
       hipMalloc((void**)&d_data, sizeof(int) * num_elements);
       hipMalloc((void**)&d_result, sizeof(int));
       hipMemcpy(d_data, data, sizeof(int) * num_elements, hipMemcpyHostToDevice);

       // Launch the kernel
       checkValues<<<1, num_elements>>>(d_data, d_result);
       hipDeviceSynchronize();

       // Copy the result back to the host
       hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

       if (result) {
               printf("All values in the thread warp are non-zero\n");
       } else {
               printf("Not all values in the thread warp are non-zero\n");
       }

       // Cleanup
       hipFree(d_data);
       hipFree(d_result);

       return 0;
}

