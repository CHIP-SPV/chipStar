#include "hip/hip_runtime.h"
// System includes
#include <stdio.h>
#include <assert.h>

// CUDA runtime
#include <hip/hip_runtime.h>
// #include <hip/hip_runtime_api.h>

// Helper functions and utilities to work with CUDA
#include <helper_functions.h>
#include <hip/hip_runtime_api.h>

#include <stdio.h>

__global__ void checkValues(int* data, int* result) {
       int tid = threadIdx.x;
       int value = data[tid];

       // Check if any value in the thread warp is greater than 5
       int any_greater_than_5 = (__any(value > 5)) ? 1 : 0;

       if (tid == 0) {
               *result = any_greater_than_5;
       }

}

int main() {
       int num_elements = 8;
       int data[] = {1, 2, 3, 4, 5, 6, 7, 8};
       int result;
       int* d_data;
       int* d_result;

       // Allocate and copy data to the device
       hipMalloc((void**)&d_data, sizeof(int) * num_elements);
       hipMalloc((void**)&d_result, sizeof(int));
       hipMemcpy(d_data, data, sizeof(int) * num_elements, hipMemcpyHostToDevice);

       // Launch the kernel
       checkValues<<<1, num_elements>>>(d_data, d_result);
       hipDeviceSynchronize();

       // Copy the result back to the host
       hipMemcpy(&result, d_result, sizeof(int), hipMemcpyDeviceToHost);

       if (result) {
               printf("At least one value in the thread warp is greater than 5\n");
       } else {
               printf("No value in the thread warp is greater than 5\n");
       }

       // Cleanup
       hipFree(d_data);
       hipFree(d_result);

       return 0;
}
