// Test an unsupported device function.

#include <hip/hip_runtime.h>
__global__ void test(unsigned *Mask) {
  *Mask = __activemask();
}

int main() {
  unsigned *MaskD;
  hipMalloc(&MaskD, sizeof(unsigned));
  test<<<1, 1>>>(MaskD);
  return 0;
}
